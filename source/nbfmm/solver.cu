#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/solver.cu
/// @brief   The implementation of the FMM solver
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <nbfmm/solver.hpp>

//  The namespace NBFMM
namespace nbfmm {

// Default constructor
Solver::Solver(
    const int            num_level,
    const int            max_num_particle,
    const float4         position_limits
) : num_level_(num_level),
    base_size_(1 << (num_level-1)),
    max_num_particle_(max_num_particle),
    position_limits_(position_limits) {
  assert(num_level > 0);
  assert(max_num_particle > 0);
  assert(position_limits.x < position_limits.z);
  assert(position_limits.y < position_limits.w);

  hipMalloc(&gpuptr_position_,      max_num_particle_ * sizeof(float2));
  hipMalloc(&gpuptr_effect_,        max_num_particle_ * sizeof(float2));
  hipMalloc(&gpuptr_weight_,        max_num_particle_ * sizeof(float));
  hipMalloc(&gpuptr_index_,         max_num_particle_ * sizeof(int2));
  hipMalloc(&gpuptr_perm_,          max_num_particle_ * sizeof(int));
  hipMalloc(&gpuptr_head_,          (base_size_ * base_size_ + 1) * sizeof(int));
  hipMalloc(&gpuptr_cell_position_, base_size_ * base_size_ * num_level_ * sizeof(float2));
  hipMalloc(&gpuptr_cell_effect_,   base_size_ * base_size_ * num_level_ * sizeof(float2));
  hipMalloc(&gpuptr_cell_weight_,   base_size_ * base_size_ * num_level_ * sizeof(float));
}

// Default destructor
Solver::~Solver() {
  hipFree(gpuptr_position_);
  hipFree(gpuptr_effect_);
  hipFree(gpuptr_weight_);
  hipFree(gpuptr_index_);
  hipFree(gpuptr_perm_);
  hipFree(gpuptr_head_);
  hipFree(gpuptr_cell_position_);
  hipFree(gpuptr_cell_effect_);
  hipFree(gpuptr_cell_weight_);
}

}  // namespace nbfmm
