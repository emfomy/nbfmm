#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/solver.cu
/// @brief   The implementation of the FMM solver
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <nbfmm/solver.hpp>

//  The namespace NBFMM
namespace nbfmm {

// Default constructor
Solver::Solver(
    const int            num_level,
    const int            max_num_particle,
    const float4         position_limits,
    const KernelFunction kernel_function
) : num_level_(num_level),
    base_size_(1 << (num_level-1)),
    max_num_particle_(max_num_particle),
    position_limits_(position_limits),
    kernel_function_(kernel_function) {
  hipMalloc(&gpuptr_position, max_num_particle_ * sizeof(float2));
  hipMalloc(&gpuptr_effect,   max_num_particle_ * sizeof(float2));
  hipMalloc(&gpuptr_weight,   max_num_particle_ * sizeof(float));
  hipMalloc(&gpuptr_index,    max_num_particle_ * sizeof(int));
  hipMalloc(&gpuptr_perm,     max_num_particle_ * sizeof(int));
  hipMalloc(&gpuptr_head,     (base_size_*base_size_+1) * sizeof(int));
  hipMalloc3D(&pitchedptr_multipole, make_hipExtent(base_size_*sizeof(float),  base_size_, num_level_));
  hipMalloc3D(&pitchedptr_local,     make_hipExtent(base_size_*sizeof(float2), base_size_, num_level_));
}

// Default destructor
Solver::~Solver() {
  hipFree(gpuptr_position);
  hipFree(gpuptr_effect);
  hipFree(gpuptr_weight);
  hipFree(gpuptr_index);
  hipFree(gpuptr_perm);
  hipFree(gpuptr_head);
  hipFree(gpuptr_multipole);
  hipFree(gpuptr_local);
}

}  // namespace nbfmm
