#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/solver.cu
/// @brief   The implementation of the FMM solver
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <nbfmm/solver.hpp>

//  The namespace NBFMM
namespace nbfmm {

// Default constructor
Solver::Solver(
    const int            num_level,
    const int            max_num_particle,
    const float4         position_limits,
    const KernelFunction kernel_function
) : num_level_(num_level),
    base_size_(1 << (num_level-1)),
    max_num_particle_(max_num_particle),
    position_limits_(position_limits),
    kernel_function_(kernel_function) {
  hipMalloc(&gpuptr_position_, max_num_particle_ * sizeof(float2));
  hipMalloc(&gpuptr_effect_,   max_num_particle_ * sizeof(float2));
  hipMalloc(&gpuptr_weight_,   max_num_particle_ * sizeof(float));
  hipMalloc(&gpuptr_index_,    max_num_particle_ * sizeof(int));
  hipMalloc(&gpuptr_perm_,     max_num_particle_ * sizeof(int));
  hipMalloc(&gpuptr_head_,     (base_size_*base_size_+1) * sizeof(int));
  hipMalloc3D(&pitchedptr_multipole_, make_hipExtent(base_size_*sizeof(float),  base_size_, num_level_));
  hipMalloc3D(&pitchedptr_local_,     make_hipExtent(base_size_*sizeof(float2), base_size_, num_level_));
}

// Default destructor
Solver::~Solver() {
  hipFree(gpuptr_position_);
  hipFree(gpuptr_effect_);
  hipFree(gpuptr_weight_);
  hipFree(gpuptr_index_);
  hipFree(gpuptr_perm_);
  hipFree(gpuptr_head_);
  hipFree(gpuptr_multipole_);
  hipFree(gpuptr_local_);
}

}  // namespace nbfmm
