#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/display/stars/update.cu
/// @brief   Update the stars
///
/// @author  Mu Yang       <emfomy@gmail.com>
/// @author  Yung-Kang Lee <blasteg@gmail.com>
/// @author  Da-Wei Chang  <davidzan830@gmail.com>
///

#include <nbfmm/display/stars.hpp>
#include <nbfmm/utility.hpp>

/// @addtogroup impl_display
/// @{

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Update the stars
///
/// @param[in]   num_star      the number of stars.
/// @param[in]   tick          the step size in time.
/// @param[out]  position_cur  the current star positions.
/// @param[out]  position_pre  the previous star positions.
/// @param[out]  acceleration  the accelerations.
///

__global__ void updateDevice(
    const int   num_star,
    const float tick,
    float2*     position_cur,
    float2*     position_pre,
    float2*     acceleration
) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if ( idx >= num_star ) {
    return;
  }

  auto position = position_cur[idx];
  position_cur[idx] = 2 * position - position_pre[idx] + acceleration[idx] * tick * tick;
  position_pre[idx] = position;
}

/// @}

// Update the stars
void nbfmm::Stars::update() {
  solver_.solve(num_star_, gpuptr_position_cur_, gpuptr_weight_, gpuptr_acceleration_);

  const int block_dim = kMaxBlockDim;
  const int grid_dim  = ((num_star_-1)/block_dim)+1;
  updateDevice<<<grid_dim, block_dim>>>(num_star_, tick_, gpuptr_position_cur_, gpuptr_position_pre_, gpuptr_acceleration_);

  prune();
}
