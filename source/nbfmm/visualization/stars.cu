#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/display/stars.cu
/// @brief   The implementation of stars class
///
/// @author  Mu Yang <emfomy@gmail.com>
/// @author  Yung-Kang Lee <blasteg@gmail.com>
/// @author  Da-Wei Chang <davidzan830@gmail.com>
///

/// @cond

#include <nbfmm/display.hpp>
#include <cstdint>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <nbfmm/utility.hpp>
#include <nbfmm/model.hpp>
#include <thrust/device_ptr.h>
#include <thrust/find.h>
#include <thrust/sort.h>

__global__ void update_kernel(int num_star,float2* gpuptr_position_cur,float2* gpuptr_position_pre,float2* gpuptr_acceleration, float tick)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float2 cur_position = gpuptr_position_cur[idx];

    if ( idx>=num_star )
    {
     return;
    }

    gpuptr_position_cur[idx] = 2*cur_position - gpuptr_position_pre[idx] + tick*tick*gpuptr_acceleration[idx];
    gpuptr_position_pre[idx] = cur_position;
}

__global__ void visualize_kernel(int num_star,float2* gpuptr_position_cur, uint8_t *board,int width, int height,float* gpuptr_weight,float size_scale,float4 display_limits)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx>=num_star )
    {
     return;
    }
    float widthUnit=(display_limits.z - display_limits.x) /width;
    float heightUnit=(display_limits.w - display_limits.y) /height;
    int pixx=floor((gpuptr_position_cur[idx].x-display_limits.x)/widthUnit);
    int pixy=height-1-floor((gpuptr_position_cur[idx].y-display_limits.y)/heightUnit);
    int size =floor(gpuptr_weight[idx]/size_scale)+1;
    if ( 0 <= pixx && pixx < width && 0 <= pixy && pixy <= height )
    {
      if (size>=1)
      {
        board[pixy*width+pixx]=255;
      }
      if (size>=2)
      {
        if (pixy>0)
          board[(pixy-1)*width+pixx]=255;
        if (pixy<height-1)
          board[(pixy+1)*width+pixx]=255;
        if (pixx>0)
          board[pixy*width+(pixx-1)]=255;
        if (pixx<width-1)
          board[pixy*width+(pixx+1)]=255;
      }
      if (size>=3)
      {
        if (pixy>0 && pixx>0)
          board[(pixy-1)*width+(pixx-1)]=255;
        if (pixy<height-1 && pixx>0)
          board[(pixy+1)*width+(pixx-1)]=255;
        if (pixy>0 && pixx<width-1)
          board[(pixy-1)*width+(pixx+1)]=255;
        if (pixy<height-1 && pixx<width-1)
          board[(pixy+1)*width+(pixx+1)]=255;
      }
      if (size>=4)
      {
        if (pixy>1)
          board[(pixy-2)*width+pixx]=255;
        if (pixy<height-2)
          board[(pixy+2)*width+pixx]=255;
        if (pixx>1)
          board[pixy*width+(pixx-2)]=255;
        if (pixx<width-2)
          board[pixy*width+(pixx+2)]=255;
      }
      if (size>=5)
      {
        if (pixy>1 && pixx>0)
          board[(pixy-2)*width+(pixx-1)]=255;
        if (pixy>1 && pixx<width-1)
          board[(pixy-2)*width+(pixx+1)]=255;
        if (pixy<height-2 && pixx>0)
          board[(pixy+2)*width+(pixx-1)]=255;
        if (pixy<height-2 && width-1)
          board[(pixy+2)*width+(pixx+1)]=255;
        if (pixy>0 && pixx<width-2)
          board[(pixy-1)*width+(pixx+2)]=255;
        if (pixy<height-1 && pixx<width-2)
          board[(pixy+1)*width+(pixx+2)]=255;
        if (pixy<height-1 && pixx>1)
          board[(pixy+1)*width+(pixx-2)]=255;
        if (pixy>0 && pixx>1)
          board[(pixy-1)*width+(pixx-2)]=255;
      }
      if (size>=6)
      {
        if (pixy>1 && pixx>1)
          board[(pixy-2)*width+(pixx-2)]=255;
        if (pixy<height-2 && pixx>2)
          board[(pixy+2)*width+(pixx-2)]=255;
        if (pixy>1 && pixx<width-2)
          board[(pixy-2)*width+(pixx+2)]=255;
        if (pixy<height-2 && pixx<width-2)
          board[(pixy+2)*width+(pixx+2)]=255;
      }
    }
}

__global__ void setup_kernel(hiprandState *state, int num_star)
{

  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  if ( idx>=num_star )
  {
   return;
  }
  hiprand_init(5678, idx, 0, &state[idx]);
}

__global__ void deletion_check_kernel(int num_star, float2* gpuptr_position_cur, float4 position_limits, int* elimination)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx>=num_star )
    {
     return;
    }
    if (gpuptr_position_cur[idx].x<position_limits.x || gpuptr_position_cur[idx].y<position_limits.y ||gpuptr_position_cur[idx].x>position_limits.z ||gpuptr_position_cur[idx].y>position_limits.w)
    {
      elimination[idx] = 1;
    } else {
      elimination[idx] = 0;
    }
}

void nbfmm::Stars::initialize()
{
  auto position_limits = position_limits_;

  // const float2 center_position = (make_float2(position_limits.x, position_limits.y) +
  //                                 make_float2(position_limits.z, position_limits.w)) / 2;
  // const float width  = (position_limits.z - position_limits.x)/2;
  // const float height = (position_limits.w - position_limits.y)/2;

  // nbfmm::generateModelRectangle(
  //     num_star_, center_position, width, height, 6.0f, tick_, gpuptr_position_cur_, gpuptr_position_pre_, gpuptr_weight_
  // );

  const int n1 = 5;
  const int n2 = 3;
  const float mu1 = float(n1) / (n1+n2);
  const float mu2 = float(n2) / (n1+n2);

  const float2 center_position1 = (make_float2(position_limits.x, position_limits.y) * (3*mu1+2*mu2) +
                                   make_float2(position_limits.z, position_limits.w) * (3*mu1+4*mu2)) / 6;
  const float2 center_position2 = (make_float2(position_limits.z, position_limits.w) * (3*mu2+2*mu1) +
                                   make_float2(position_limits.x, position_limits.y) * (3*mu2+4*mu1)) / 6;
  const float radius = (position_limits.w - position_limits.y)/16;

  // nbfmm::generateModelDisk(
  //     num_star_, center_position1, radius, 3.0f, tick_, gpuptr_position_cur_, gpuptr_position_pre_, gpuptr_weight_
  // );

  nbfmm::generateModelDoubleDisk(
      num_star_*mu1, num_star_*mu2, center_position1, center_position2, radius*mu1, radius*mu2, 3.0f, tick_,
      gpuptr_position_cur_, gpuptr_position_pre_, gpuptr_weight_
  );

  // nbfmm::generateModelDoubleDiskCenter(
  //     num_star_*mu1, num_star_*mu2, center_position1, center_position2, radius*mu1, radius*mu2, 1.0f,
  //     num_star_*mu1, num_star_*mu1, tick_, gpuptr_position_cur_, gpuptr_position_pre_, gpuptr_weight_
  // );
}
//update
void nbfmm::Stars::update()
{
  solver_.solve(num_star_, gpuptr_position_cur_, gpuptr_weight_, gpuptr_acceleration_);
  const int kNumThread_pointwise = 1024;
  const int kNumBlock_pointwise  = ((num_star_-1)/kNumThread_pointwise)+1;
  update_kernel<<<kNumBlock_pointwise,kNumThread_pointwise>>>(num_star_,gpuptr_position_cur_,gpuptr_position_pre_,gpuptr_acceleration_,tick_);

  deletion_check();
}

void nbfmm::Stars::display(uint8_t *board)
{
  auto width = width_;
  auto height = height_;
  auto size_scale = size_scale_;
  auto display_limits = display_limits_;
  hipMemset(board, 0, width*height);
  hipMemset(board+width*height, 128, width*height/2);
  const int kNumThread_pointwise = 1024;
  const int kNumBlock_pointwise  = ((num_star_-1)/kNumThread_pointwise)+1;
  visualize_kernel<<<kNumBlock_pointwise,kNumThread_pointwise>>>(num_star_,gpuptr_position_cur_,board,width,height,gpuptr_weight_, size_scale, display_limits);

  deletion_check();
}

void nbfmm::Stars::deletion_check()
{
  auto position_limits = position_limits_;
  int* elimination;
  hipMalloc(&elimination, sizeof(int)*num_star_);
  const int kNumThread_pointwise = 1024;
  const int kNumBlock_pointwise  = ((num_star_-1)/kNumThread_pointwise)+1;
  deletion_check_kernel<<<kNumBlock_pointwise,kNumThread_pointwise>>>(num_star_, gpuptr_position_cur_, position_limits, elimination);

  // zip cur and pre position
  thrust::device_ptr<float2> thrust_position_cur(gpuptr_position_cur_);
  thrust::device_ptr<float2> thrust_position_pre(gpuptr_position_pre_);
  thrust::device_ptr<float>  thrust_weight(gpuptr_weight_);
  thrust::zip_iterator<thrust::tuple<thrust::device_ptr<float2>, thrust::device_ptr<float2>, thrust::device_ptr<float>>>
  position_iter(thrust::make_tuple(thrust_position_cur, thrust_position_pre, thrust_weight));

  // Perform elimination
  thrust::device_ptr<int>    thrust_elimination(elimination);
  thrust::sort_by_key(thrust_elimination, thrust_elimination + num_star_, position_iter);
  num_star_ = thrust::find(thrust_elimination, thrust_elimination + num_star_, 1) - thrust_elimination;

  hipFree(elimination);
}

/// @endcond
