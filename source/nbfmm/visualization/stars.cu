#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/visualization/stars.cu
/// @brief   The implementation of stars class
///
/// @author  Mu Yang <emfomy@gmail.com>
/// @author  Yung-Kang Lee <blasteg@gmail.com>
/// @author  Da-Wei Chang <davidzan830@gmail.com>
///

/// @cond

#include <nbfmm/visualization.hpp>
#include <cstdint>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <nbfmm/utility.hpp>
#include <nbfmm/model.hpp>
#include <thrust/device_ptr.h>
#include <thrust/find.h>
#include <thrust/sort.h>

__global__ void update_kernel(int n_star,float2* gpu_star_position_cur,float2* gpu_star_position_pre,float2* gpu_star_acceleration, float dt)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float2 cur_position = gpu_star_position_cur[idx];

    if ( idx>=n_star )
    {
     return;
    }

    gpu_star_position_cur[idx] = 2*cur_position - gpu_star_position_pre[idx] + dt*dt*gpu_star_acceleration[idx];
    gpu_star_position_pre[idx] = cur_position;
}

__global__ void visualize_kernel(int n_star,float2* gpu_star_position_cur, uint8_t *board,int width, int height,float* gpu_star_weight,float size_th,float4 visualization_limits)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx>=n_star )
    {
     return;
    }
    float widthUnit=(visualization_limits.z - visualization_limits.x) /width;
    float heightUnit=(visualization_limits.w - visualization_limits.y) /height;
    int pixx=floor((gpu_star_position_cur[idx].x-visualization_limits.x)/widthUnit);
    int pixy=height-1-floor((gpu_star_position_cur[idx].y-visualization_limits.y)/heightUnit);
    int size =floor(gpu_star_weight[idx]/size_th)+1;
    if ( 0 <= pixx && pixx < width && 0 <= pixy && pixy <= height )
    {
      if (size>=1)
      {
        board[pixy*width+pixx]=255;
      }
      if (size>=2)
      {
        if (pixy>0)
          board[(pixy-1)*width+pixx]=255;
        if (pixy<height-1)
          board[(pixy+1)*width+pixx]=255;
        if (pixx>0)
          board[pixy*width+(pixx-1)]=255;
        if (pixx<width-1)
          board[pixy*width+(pixx+1)]=255;
      }
      if (size>=3)
      {
        if (pixy>0 && pixx>0)
          board[(pixy-1)*width+(pixx-1)]=255;
        if (pixy<height-1 && pixx>0)
          board[(pixy+1)*width+(pixx-1)]=255;
        if (pixy>0 && pixx<width-1)
          board[(pixy-1)*width+(pixx+1)]=255;
        if (pixy<height-1 && pixx<width-1)
          board[(pixy+1)*width+(pixx+1)]=255;
      }
      if (size>=4)
      {
        if (pixy>1)
          board[(pixy-2)*width+pixx]=255;
        if (pixy<height-2)
          board[(pixy+2)*width+pixx]=255;
        if (pixx>1)
          board[pixy*width+(pixx-2)]=255;
        if (pixx<width-2)
          board[pixy*width+(pixx+2)]=255;
      }
      if (size>=5)
      {
        if (pixy>1 && pixx>0)
          board[(pixy-2)*width+(pixx-1)]=255;
        if (pixy>1 && pixx<width-1)
          board[(pixy-2)*width+(pixx+1)]=255;
        if (pixy<height-2 && pixx>0)
          board[(pixy+2)*width+(pixx-1)]=255;
        if (pixy<height-2 && width-1)
          board[(pixy+2)*width+(pixx+1)]=255;
        if (pixy>0 && pixx<width-2)
          board[(pixy-1)*width+(pixx+2)]=255;
        if (pixy<height-1 && pixx<width-2)
          board[(pixy+1)*width+(pixx+2)]=255;
        if (pixy<height-1 && pixx>1)
          board[(pixy+1)*width+(pixx-2)]=255;
        if (pixy>0 && pixx>1)
          board[(pixy-1)*width+(pixx-2)]=255;
      }
      if (size>=6)
      {
        if (pixy>1 && pixx>1)
          board[(pixy-2)*width+(pixx-2)]=255;
        if (pixy<height-2 && pixx>2)
          board[(pixy+2)*width+(pixx-2)]=255;
        if (pixy>1 && pixx<width-2)
          board[(pixy-2)*width+(pixx+2)]=255;
        if (pixy<height-2 && pixx<width-2)
          board[(pixy+2)*width+(pixx+2)]=255;
      }
    }
}

__global__ void setup_kernel(hiprandState *state, int n_star)
{

  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  if ( idx>=n_star )
  {
   return;
  }
  hiprand_init(5678, idx, 0, &state[idx]);
}

__global__ void deletion_check_kernel(int n_star, float2* gpu_star_position_cur, float4 position_limit, int* elimination)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx>=n_star )
    {
     return;
    }
    if (gpu_star_position_cur[idx].x<position_limit.x || gpu_star_position_cur[idx].y<position_limit.y ||gpu_star_position_cur[idx].x>position_limit.z ||gpu_star_position_cur[idx].y>position_limit.w)
    {
      elimination[idx] = 1;
    } else {
      elimination[idx] = 0;
    }
}
// Constructor
Stars::Stars(int nStar, int FPS)
 : n_star(nStar), FPS(FPS), dt(0.05f/FPS)
 {
  assert(nStar>0);
  hipMalloc(&gpu_star_position_cur,  n_star*sizeof(float2));
  hipMalloc(&gpu_star_position_pre,  n_star*sizeof(float2));
  hipMalloc(&gpu_star_acceleration,  n_star*sizeof(float2));
  hipMalloc(&gpu_star_weight,n_star*sizeof(float));
 }
//Destructor
Stars::~Stars()
{
  hipFree(gpu_star_position_cur);
  hipFree(gpu_star_position_pre);
  hipFree(gpu_star_acceleration);
  hipFree(gpu_star_weight);
}

void Stars::initialize(float4 position_limit)
{
  // const int n1 = 5;
  // const int n2 = 3;
  // const float mu1 = float(n1) / (n1+n2);
  // const float mu2 = float(n2) / (n1+n2);

  const float2 center_position = (make_float2(position_limit.x, position_limit.y) +
                                  make_float2(position_limit.z, position_limit.w)) / 2;
  const float width  = (position_limit.z - position_limit.x)/2;
  const float height = (position_limit.w - position_limit.y)/2;

  // const float2 center_position1 = (make_float2(position_limit.x, position_limit.y) * (3*mu1+2*mu2) +
  //                                  make_float2(position_limit.z, position_limit.w) * (3*mu1+4*mu2)) / 6;
  // const float2 center_position2 = (make_float2(position_limit.z, position_limit.w) * (3*mu2+2*mu1) +
  //                                  make_float2(position_limit.x, position_limit.y) * (3*mu2+4*mu1)) / 6;
  // const float radius = (position_limit.w - position_limit.y)/16;

  nbfmm::generateModelRectangle(
      n_star, center_position, width, height, 6.0f, dt, gpu_star_position_cur, gpu_star_position_pre, gpu_star_weight
  );

  // nbfmm::generateModelDisk(
  //     n_star, center_position1, radius, 3.0f, dt, gpu_star_position_cur, gpu_star_position_pre, gpu_star_weight
  // );

  // nbfmm::generateModelDoubleDisk(
  //     n_star*mu1, n_star*mu2, center_position1, center_position2, radius*mu1, radius*mu2, 3.0f, dt,
  //     gpu_star_position_cur, gpu_star_position_pre, gpu_star_weight
  // );

  // nbfmm::generateModelDoubleDiskCenter(
  //     n_star*mu1, n_star*mu2, center_position1, center_position2, radius*mu1, radius*mu2, 1.0f,
  //     n_star*mu1, n_star*mu1, dt, gpu_star_position_cur, gpu_star_position_pre, gpu_star_weight
  // );
}
//update
void Stars::update()
{
  const int kNumThread_pointwise = 1024;
  const int kNumBlock_pointwise  = ((n_star-1)/kNumThread_pointwise)+1;
  update_kernel<<<kNumBlock_pointwise,kNumThread_pointwise>>>(n_star,gpu_star_position_cur,gpu_star_position_pre,gpu_star_acceleration,dt);
}

void Stars::visualize(int width, int height, uint8_t *board,float size_th,float4  visualization_limits)
{
  hipMemset(board, 0, width*height);
  hipMemset(board+width*height, 128, width*height/2);
  const int kNumThread_pointwise = 1024;
  const int kNumBlock_pointwise  = ((n_star-1)/kNumThread_pointwise)+1;
  visualize_kernel<<<kNumBlock_pointwise,kNumThread_pointwise>>>(n_star,gpu_star_position_cur,board,width,height,gpu_star_weight, size_th, visualization_limits);
}

void Stars::deletion_check(float4 position_limit)
{
  int* elimination;
  hipMalloc(&elimination, sizeof(int)*n_star);
  const int kNumThread_pointwise = 1024;
  const int kNumBlock_pointwise  = ((n_star-1)/kNumThread_pointwise)+1;
  deletion_check_kernel<<<kNumBlock_pointwise,kNumThread_pointwise>>>(n_star, gpu_star_position_cur, position_limit, elimination);

  // zip cur and pre position
  thrust::device_ptr<float2> thrust_position_cur(gpu_star_position_cur);
  thrust::device_ptr<float2> thrust_position_pre(gpu_star_position_pre);
  thrust::device_ptr<float>  thrust_weight(gpu_star_weight);
  thrust::zip_iterator<thrust::tuple<thrust::device_ptr<float2>, thrust::device_ptr<float2>, thrust::device_ptr<float>>>
  position_iter(thrust::make_tuple(thrust_position_cur, thrust_position_pre, thrust_weight));

  // Perform elimination
  thrust::device_ptr<int>    thrust_elimination(elimination);
  thrust::sort_by_key(thrust_elimination, thrust_elimination + n_star, position_iter);
  n_star = thrust::find(thrust_elimination, thrust_elimination + n_star, 1) - thrust_elimination;

  hipFree(elimination);
}

/// @endcond
