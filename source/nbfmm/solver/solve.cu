#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/solver/solve.cu
/// @brief   Solve system
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <nbfmm/solver.hpp>

//  The namespace NBFMM
namespace nbfmm {

// Solve system
void Solver::solve(
    const int     num_particle,
    const float2* gpuptr_position_origin,
    const float*  gpuptr_weight_origin,
    float2*       gpuptr_effect_origin
) {
  assert(num_particle >= 0 && num_particle <= max_num_particle_);

  predo(num_particle, gpuptr_position_origin, gpuptr_weight_origin);
  p2p(num_particle);
#pragma warning
  // p2m(num_particle);
  // m2m();
  // m2l();
  // l2l();
  // l2p(num_particle);
  postdo(num_particle, gpuptr_effect_origin);
}

}  // namespace nbfmm
