#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/solver/l2p.cu
/// @brief   Compute local to particle
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <nbfmm/solver.hpp>

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Compute local to local
///
/// @param[in]      num_level    the number of cell levels.
/// @param[in]      base_size    the number of girds in the base level per side.
/// @param[in/out]  cell_effect  the cell effects.
///
__global__ void l2lDevice(
    const int num_level,
    const int base_size,
    float2*   cell_effect
) {
  const int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  int idx   = idx_x + idx_y * base_size + num_level * base_size * base_size;
  for ( int cell_size = base_size; cell_size >= 2; cell_size /= 2 ) {
    int idx_above = idx;
    idx -= base_size * base_size;
    if ( (idx_x % cell_size == 0) && (idx_y % cell_size == 0) ) {
      cell_effect[idx].x             += cell_effect[idx_above].x;
      cell_effect[idx+1].x           += cell_effect[idx_above].x;
      cell_effect[idx+base_size].x   += cell_effect[idx_above].x;
      cell_effect[idx+base_size+1].x += cell_effect[idx_above].x;
      cell_effect[idx].y             += cell_effect[idx_above].y;
      cell_effect[idx+1].y           += cell_effect[idx_above].y;
      cell_effect[idx+base_size].y   += cell_effect[idx_above].y;
      cell_effect[idx+base_size+1].y += cell_effect[idx_above].y;
    }
    __syncthreads();
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Compute local to particle
///
/// @param[in]      num_particle  the number of particles.
/// @param[in]      base_size     the number of girds in the base level per side.
/// @param[in]      index         the particle cell indices.
/// @param[in]      cell_effect   the cell effects.
/// @param[in/out]  effect        the particle effects.
///
__global__ void l2pDevice(
    const int     num_particle,
    const int     base_size,
    const int2*   index,
    const float2* cell_effect,
    float2*       effect
) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if ( idx >= num_particle ) {
    return;
  }
  const int cell_idx = index[idx].x + index[idx].y * base_size;
  effect[idx].x += cell_effect[cell_idx].x;
  effect[idx].y += cell_effect[cell_idx].y;
}

//  The namespace NBFMM
namespace nbfmm {

// L2P
void Solver::l2p( const int num_particle ) {
  {
    if ( num_level_ <= 1 ) {
      return;
    }
    const int block_dim_side = (base_size_ < kMaxBlockDim) ? base_size_ : kMaxBlockDim;
    const int grid_dim_side  = (base_size_ < kMaxBlockDim) ? 1 : (base_size_ / block_dim_side);
    assert(grid_dim_side <= kMaxGridDim);
    const dim3 block_dim(block_dim_side, block_dim_side);
    const dim3 grid_dim(grid_dim_side, grid_dim_side);
    l2lDevice<<<block_dim, grid_dim>>>(num_level_, base_size_, gpuptr_cell_effect_);
  }
  {
    if ( num_level_ <= 0 ) {
      return;
    }
    const int block_dim = kMaxBlockDim;
    const int grid_dim  = ((num_particle-1)/block_dim)+1;
    assert(grid_dim <= kMaxGridDim);
    l2pDevice<<<block_dim, grid_dim>>>(num_particle, base_size_, gpuptr_index_, gpuptr_cell_effect_, gpuptr_effect_);
  }
}

}  // namespace nbfmm
