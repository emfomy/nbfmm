#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/model/double_disk_center.cu
/// @brief   The implementation of the generator for double disk shape particles with a large particle at each center
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <nbfmm/model.hpp>
#include <cmath>
#include <nbfmm/core/kernel_function.hpp>
#include <nbfmm/utility.hpp>

/// @addtogroup impl_model
/// @{

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Generate double disk shape particles with a large particle at each center
///
/// @param[in]   num_particle       the number of particles.
/// @param[in]   offset              the offset of previous particle positions.
/// @param[out]  position_previous  the previous particle positions.
///
__global__ void generateDoubleDiskCenterDevice(
    const int num_particle,
    float2    offset,
    float2*   position_previous
) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if ( idx >= num_particle ) {
    return;
  }
  position_previous[idx] += offset;
}

/// @}

// Generate double disk shape particles with a large particle at each center
void nbfmm::model::generateDoubleDiskCenter(
    const int    num_particle1,
    const int    num_particle2,
    const float2 center_position1,
    const float2 center_position2,
    const float  radius1,
    const float  radius2,
    const float  weight,
    const float  center_weight1,
    const float  center_weight2,
    const float  eccentricity,
    const float  tick,
    float2*      gpuptr_position_current,
    float2*      gpuptr_position_previous,
    float*       gpuptr_weight_current
) {
  generateDiskCenter(num_particle1, center_position1, radius1, weight, center_weight1, tick,
                    gpuptr_position_current, gpuptr_position_previous, gpuptr_weight_current);
  generateDiskCenter(num_particle2, center_position2, radius2, weight, center_weight2, tick,
                    gpuptr_position_current+num_particle1, gpuptr_position_previous+num_particle1,
                    gpuptr_weight_current+num_particle1);

  const float weight1 = weight * (num_particle1-1) + center_weight1;
  const float weight2 = weight * (num_particle2-1) + center_weight2;

  const float2 effect1 = kernelFunction(center_position1, center_position2, weight2);
  const float2 effect2 = kernelFunction(center_position2, center_position1, weight1);

  float2 distance = center_position1 - center_position2;
  float r = sqrt(distance.x * distance.x + distance.y * distance.y);
  float a1 = sqrt(effect1.x * effect1.x + effect1.y * effect1.y);
  float a2 = sqrt(effect2.x * effect2.x + effect2.y * effect2.y);
  float r1 = r * weight2 / (weight1 + weight2);
  float r2 = r * weight1 / (weight1 + weight2);

  float2 offset1;
  offset1.x = -effect1.y; offset1.y = effect1.x;
  offset1 *= sqrt(r1/a1) * tick / exp2(eccentricity);
  offset1 -= effect1 * tick * tick * eccentricity;

  float2 offset2;
  offset2.x = -effect2.y; offset2.y = effect2.x;
  offset2 *= sqrt(r2/a2) * tick / exp2(eccentricity);
  offset2 -= effect2 * tick * tick * eccentricity;

  generateDoubleDiskCenterDevice<<<kMaxBlockDim, ((num_particle1-1)/kMaxBlockDim)+1>>>(
      num_particle1, offset1, gpuptr_position_previous
  );
  generateDoubleDiskCenterDevice<<<kMaxBlockDim, ((num_particle2-1)/kMaxBlockDim)+1>>>(
      num_particle2, offset2, gpuptr_position_previous+num_particle1
  );
}
