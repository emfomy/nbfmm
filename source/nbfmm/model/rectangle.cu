#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/model/rectangle.cu
/// @brief   The implementation of rectangle shape generator.
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <nbfmm/model.hpp>
#include <cmath>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <nbfmm/core/kernel_function.hpp>
#include <nbfmm/utility.hpp>

/// @addtogroup impl_model
/// @{

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Generate rectangle shape particles
///
/// @param[in]   num_particle       the number of particles.
/// @param[in]   center_position    the center position.
/// @param[in]   width              the width.
/// @param[in]   height             the height.
/// @param[in]   max_weight         the maximum weight.
/// @param[in]   tick               the step size in time.
/// @param[out]  position_current   the current particle positions.
/// @param[out]  position_previous  the previous particle positions.
/// @param[out]  weight             the particle weights.
///
__global__ void generateRectangleDevice(
    const int     num_particle,
    const float2  center_position,
    const float   width,
    const float   height,
    const float   max_weight,
    const float   tick,
    float2*       position_current,
    float2*       position_previous,
    float*        weight
) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if ( idx >= num_particle ) {
    return;
  }

  hiprandState s;
  hiprand_init(0, idx, 0, &s);

  const float2 position = center_position + make_float2((hiprand_uniform(&s) - 0.5f) * width,
                                                        (hiprand_uniform(&s) - 0.5f) * height);
  position_current[idx]  = position;
  position_previous[idx] = position;
  weight[idx]            = max_weight * hiprand_uniform(&s);
}

/// @}

// Generate rectangle shape particles
void nbfmm::model::generateRectangle(
    const int     num_particle,
    const float2  center_position,
    const float   width,
    const float   height,
    const float   max_weight,
    const float   tick,
    float2*       gpuptr_position_current,
    float2*       gpuptr_position_previous,
    float*        gpuptr_weight
) {
  assert( num_particle > 0 );
  assert( width > 0 );
  assert( height > 0 );
  assert( max_weight > 0 );

  const int block_dim = kMaxBlockDim;
  const int grid_dim  = ((num_particle-1)/block_dim)+1;

  generateRectangleDevice<<<grid_dim, block_dim>>>(num_particle, center_position, width, height, max_weight, tick,
                                                   gpuptr_position_current, gpuptr_position_previous, gpuptr_weight);
}
