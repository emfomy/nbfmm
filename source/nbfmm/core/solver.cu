#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/core/solver.cu
/// @brief   The implementation of the FMM solver
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <nbfmm/core/solver.hpp>

// The NBFMM namespace
namespace nbfmm {

// Default constructor
Solver::Solver(
    const int            num_level,
    const int            max_num_particle,
    const float4         position_limits
) : num_level_(num_level),
    base_dim_(1 << (num_level+1)),
    num_cell_p1_(base_dim_*base_dim_+1),
    max_num_particle_(max_num_particle),
    position_limits_(position_limits)
{
  assert(num_level >= 0);
  assert(base_dim_ < kMaxBlockDim);
  assert(max_num_particle > 0 && max_num_particle < kMaxGridDim);
  assert(position_limits.x < position_limits.z && position_limits.y < position_limits.w);

  hipMalloc(&gpuptr_position_,      max_num_particle_ * sizeof(float2));
  hipMalloc(&gpuptr_effect_,        max_num_particle_ * sizeof(float2));
  hipMalloc(&gpuptr_weight_,        max_num_particle_ * sizeof(float));
  hipMalloc(&gpuptr_index_,         max_num_particle_ * sizeof(int2));
  hipMalloc(&gpuptr_perm_,          max_num_particle_ * sizeof(int));
  hipMalloc(&gpuptr_head_,          num_cell_p1_      * sizeof(int));
#pragma warning
  if ( num_level_ < 2 ) {
    hipMalloc(&gpuptr_cell_position_, base_dim_ * base_dim_ * 2 * sizeof(float2));
    hipMalloc(&gpuptr_cell_effect_,   base_dim_ * base_dim_ * 2 * sizeof(float2));
    hipMalloc(&gpuptr_cell_weight_,   base_dim_ * base_dim_ * 2 * sizeof(float));
  } else {
    hipMalloc(&gpuptr_cell_position_, base_dim_ * base_dim_ * num_level_ * sizeof(float2));
    hipMalloc(&gpuptr_cell_effect_,   base_dim_ * base_dim_ * num_level_ * sizeof(float2));
    hipMalloc(&gpuptr_cell_weight_,   base_dim_ * base_dim_ * num_level_ * sizeof(float));
  }
  hipMalloc(&gpuptr_buffer_float2_, max_num_particle_     * sizeof(float2));
  hipMalloc(&gpuptr_buffer_int2_,   base_dim_ * base_dim_ * sizeof(int2));
}

// Default destructor
Solver::~Solver() {
  hipFree(gpuptr_position_);
  hipFree(gpuptr_effect_);
  hipFree(gpuptr_weight_);
  hipFree(gpuptr_index_);
  hipFree(gpuptr_perm_);
  hipFree(gpuptr_head_);
  hipFree(gpuptr_cell_position_);
  hipFree(gpuptr_cell_effect_);
  hipFree(gpuptr_cell_weight_);
  hipFree(gpuptr_buffer_float2_);
  hipFree(gpuptr_buffer_int2_);
}

}  // namespace nbfmm
