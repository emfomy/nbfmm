#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/core/solver/m2l.cu
/// @brief   Compute multipole to local
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <nbfmm/core.hpp>
#include <nbfmm/utility.hpp>

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Compute multipole to local
///
/// @param[in]   base_dim             the number of cells in base level per side.
/// @param[in]   level_dim            the number of cells in this level per side.
/// @param[in]   cell_size            the size of cell of this level.
/// @param[in]   cell_level_position  the cell positions of this level.
/// @param[in]   cell_level_weight    the cell weights of this level.
/// @param[out]  cell_level_effect    the cell effects of this level.
///
__global__ void m2lDevice(
    const int     base_dim,
    const int     level_dim,
    const int     cell_size,
    const float2* cell_level_position,
    const float*  cell_level_weight,
    float2*       cell_level_effect
) {
  const int target_x = threadIdx.x + blockIdx.x * blockDim.x;
  const int target_y = threadIdx.y + blockIdx.y * blockDim.y;

  if ( target_x >= level_dim || target_y >= level_dim ) {
    return;
  }

  const int parent_x = target_x & ~1;
  const int parent_y = target_y & ~1;
  const int target_idx = (target_x + target_y * base_dim) * cell_size;
  const float2 target_position = cell_level_position[target_idx];
  float2 target_effect = make_float2(0.0f, 0.0f);

  // Go through children of parent cell's neighbors
  for ( int y = parent_y-2; y < parent_y+4; ++y ) {
    if ( y >= 0 && y < level_dim ) {
      for ( int x = parent_x-2; x < parent_x+4; ++x ) {
        if ( x >= 0 && x < level_dim ) {
          // Ignore target cell's neighbors
          if ( abs(x-target_x) > 1 || abs(y-target_y) > 1 ) {
            int idx = (x + y * base_dim) * cell_size;
            target_effect += nbfmm::kernelFunction(target_position, cell_level_position[idx], cell_level_weight[idx]);
          }
        }
      }
    }
  }
  cell_level_effect[target_idx] = target_effect;
}

//  The namespace NBFMM
namespace nbfmm {

// M2L
void Solver::m2l() {
  if ( num_level_ <= 0 ) {
    return;
  }

  int level_dim = base_dim_;
  int cell_size = 1;
  for ( auto level = 0; level < num_level_; ++level, level_dim /= 2, cell_size *= 2 ) {
    const int block_dim_side = (level_dim < kMaxBlockDim) ? level_dim : kMaxBlockDim;
    const int grid_dim_side  = (level_dim < kMaxBlockDim) ? 1 : (level_dim / block_dim_side);
    const dim3 block_dim(block_dim_side, block_dim_side);
    const dim3 grid_dim(grid_dim_side, grid_dim_side);
    const int offset = level * base_dim_ * base_dim_;
    m2lDevice<<<block_dim, grid_dim>>>(base_dim_, level_dim, cell_size,
                                       gpuptr_cell_position_ + offset, gpuptr_cell_weight_ + offset, gpuptr_cell_effect_ + offset);
  }
}

}  // namespace nbfmm
