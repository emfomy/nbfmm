#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/core/solver/predo.cu
/// @brief   Do preliminary works
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <nbfmm/core.hpp>
#include <nbfmm/utility.hpp>

/// @addtogroup impl_core
/// @{

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Compute cell index of each particle
///
/// @param[in]   num_particle     the number of particles.
/// @param[in]   position_limits  the limits of positions. [x_min, y_min, x_max, y_max].
/// @param[in]   cell_size        the size of gird. [width, height].
/// @param[in]   position_origin  the original particle positions.
/// @param[out]  index            the particle cell indices.
///
__global__ void computeParticleIndex(
    const int     num_particle,
    const float4  position_limits,
    const float2  cell_size,
    const float2* position_origin,
    int2*         index
) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if ( idx >= num_particle ) {
    return;
  }
  index[idx].x = floorf((position_origin[idx].x - position_limits.x) / cell_size.x);
  index[idx].y = floorf((position_origin[idx].y - position_limits.y) / cell_size.y);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Extract heads of cell index of each cell
///
/// @param[in]   num_particle  the number of particles.
/// @param[in]   base_dim      the number of cells in the base level per side.
/// @param[in]   index         the particle cell indices.
/// @param[out]  head          the starting permutation indices of each cell.
///
__global__ void extractHead(
    const int   num_particle,
    const int   base_dim,
    const int2* index,
    int*        head
) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if ( idx > num_particle ) {
    return;
  }
  const int cell_idx      = index[idx].x   + index[idx].y   * base_dim;
  const int cell_idx_past = index[idx-1].x + index[idx-1].y * base_dim;
  if ( idx == 0 ) {
    for ( auto i = 0; i <= cell_idx; ++i ) {
      head[i] = idx;
    }
  } else if ( idx == num_particle ) {
    for ( auto i = cell_idx_past+1; i <= base_dim * base_dim; ++i ) {
      head[i] = idx;
    }
  } else {
    for ( auto i = cell_idx_past+1; i <= cell_idx; ++i ) {
      head[i] = idx;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Permute input vectors
///
/// @param[in]   num_particle     the number of particles.
/// @param[in]   perm             the particle permutation indices.
/// @param[in]   position_origin  the original particle positions.
/// @param[in]   weight_origin    the original particle weights.
/// @param[out]  position         the particle positions.
/// @param[out]  weight           the particle weights.
///
__global__ void permuteInputVector(
    const int     num_particle,
    const int*    perm,
    const float2* position_origin,
    const float*  weight_origin,
    float2*       position,
    float*        weight
) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if ( idx >= num_particle ) {
    return;
  }
  position[idx] = position_origin[perm[idx]];
  weight[idx]   = weight_origin[perm[idx]];
}

/// @}

// Solve system
void nbfmm::Solver::predo(
    const int     num_particle,
    const float2* gpuptr_position_origin,
    const float*  gpuptr_weight_origin
) {
  const int block_dim = kMaxBlockDim;
  const int grid_dim  = ((num_particle-1)/block_dim)+1;
  const float2 cell_size = make_float2((position_limits_.z - position_limits_.x) / base_dim_,
                                       (position_limits_.w - position_limits_.y) / base_dim_);
  thrust::device_ptr<int2> thrust_index(gpuptr_index_);
  thrust::device_ptr<int>  thrust_perm(gpuptr_perm_);
  thrust::device_ptr<int>  thrust_head(gpuptr_head_);

  // Compute cell index of each particle
  computeParticleIndex<<<grid_dim, block_dim>>>(num_particle, position_limits_, cell_size,
                                                gpuptr_position_origin, gpuptr_index_);

  // Fill particle permutation vector
  thrust::counting_iterator<int> count_iter(0);
  thrust::copy_n(count_iter, num_particle, thrust_perm);

  // Sort values
  thrust::sort_by_key(thrust_index, thrust_index+num_particle, thrust_perm);

  // Extract heads of cell index of each cell
  extractHead<<<grid_dim, block_dim>>>(num_particle, base_dim_, gpuptr_index_, gpuptr_head_);

  // Permute input vectors
  permuteInputVector<<<grid_dim, block_dim>>>(num_particle, gpuptr_perm_,
                                              gpuptr_position_origin, gpuptr_weight_origin, gpuptr_position_, gpuptr_weight_);
}
