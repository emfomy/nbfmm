#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/core/solver/p2m.cu
/// @brief   Compute particle to multipole
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <nbfmm/core.hpp>
#include <nbfmm/utility.hpp>

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Compute particle to multipole weighting
///
/// @param[in]   num_particle     the number of particles.
/// @param[in]   gpuptr_position  the original particle effects.
/// @param[in]   gpuptr_weight    the particle effects.
/// @param[out]  p2m_buffer       the workspace.
///
__global__ void p2m_weighting( int num_particle, float2* gpuptr_position, float* gpuptr_weight, float2* p2m_buffer ) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if ( idx>=num_particle )
  {
   return;
  }
  p2m_buffer[idx].x=gpuptr_position[idx].x*gpuptr_weight[idx];
  p2m_buffer[idx].y=gpuptr_position[idx].y*gpuptr_weight[idx];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Compute particle to multipole averaging
///
/// @param[in]   base_dim              the number of cells in the base level per side.
/// @param[out]  gpuptr_cell_position  the cell positions.
/// @param[in]   gpuptr_cell_weight    the cell positions.
///

__global__ void p2m_assigning( int base_dim, float2* gpuptr_cell_position, float* gpuptr_cell_weight,int assigning_length,int2* p2m_assigningIndex ) {
  int thread2Dpx = blockIdx.x * blockDim.x + threadIdx.x;
  int thread2Dpy = blockIdx.y * blockDim.y + threadIdx.y;
  if (thread2Dpx >= base_dim || thread2Dpy >= base_dim)

   return;
  int thread1Dp = thread2Dpy * base_dim + thread2Dpx;

  gpuptr_cell_position[thread1Dp].x=0;
  gpuptr_cell_position[thread1Dp].y=0;
  gpuptr_cell_weight[thread1Dp]=0;

  if (thread1Dp>=assigning_length)
    return;
  int index_to_assign=p2m_assigningIndex[thread1Dp].y * base_dim +p2m_assigningIndex[thread1Dp].x;
  gpuptr_cell_position[index_to_assign].x=gpuptr_cell_position[base_dim*base_dim+thread1Dp].x/gpuptr_cell_weight[base_dim*base_dim+thread1Dp];
  gpuptr_cell_position[index_to_assign].y=gpuptr_cell_position[base_dim*base_dim+thread1Dp].y/gpuptr_cell_weight[base_dim*base_dim+thread1Dp];
  gpuptr_cell_weight[index_to_assign]=gpuptr_cell_weight[base_dim*base_dim+thread1Dp];
}

//  The namespace NBFMM
namespace nbfmm {

// P2M
void Solver::p2m( const int num_particle ) {
  const dim3 kNumThread_cellwise(32,32,1);
  const dim3 kNumBlock_cellwise(((base_dim_-1)/kNumThread_cellwise.x)+1,((base_dim_-1)/kNumThread_cellwise.y)+1,1);
  const int kNumThread_pointwise = 1024;
  const int kNumBlock_pointwise  = ((num_particle-1)/kNumThread_pointwise)+1;

  float2* p2m_buffer;
  int2* p2m_assigningIndex;
  hipMalloc(&p2m_buffer,      max_num_particle_ * sizeof(float2));

  hipMalloc(&p2m_assigningIndex, base_dim_ * base_dim_*sizeof(int2));

  p2m_weighting<<<kNumBlock_pointwise,kNumThread_pointwise>>>(num_particle,gpuptr_position_,gpuptr_weight_,p2m_buffer);

  thrust::device_ptr<int2> thrust_index(gpuptr_index_);
  thrust::device_ptr<float2> thrust_position(gpuptr_position_);
  thrust::device_ptr<float> thrust_weight(gpuptr_weight_);
  thrust::device_ptr<float2> thrust_weighted(p2m_buffer);
  thrust::device_ptr<int2> thrust_assigninging(p2m_assigningIndex);
  thrust::device_ptr<float2> thrust_cellPos(gpuptr_cell_position_);
  thrust::device_ptr<float> thrust_cellWei(gpuptr_cell_weight_);


  thrust::pair<int2*,float*> p2m_dummy;

  thrust::reduce_by_key(thrust_index, thrust_index + num_particle, thrust_weighted, thrust_assigninging, thrust_cellPos+base_dim_ * base_dim_);
  p2m_dummy=thrust::reduce_by_key(thrust_index, thrust_index + num_particle, thrust_weight, thrust_assigninging, thrust_cellWei+base_dim_ * base_dim_);

  int assigning_length=p2m_dummy.second;
  p2m_assigning<<<kNumBlock_cellwise,kNumThread_cellwise>>>(base_dim_,gpuptr_cell_position_,gpuptr_cell_weight_,assigning_length,p2m_assigningIndex);

  hipFree(p2m_buffer);
  hipFree(p2m_assigningIndex);

}

}  // namespace nbfmm
