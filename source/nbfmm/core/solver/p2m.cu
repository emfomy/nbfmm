#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    source/nbfmm/core/solver/p2m.cu
/// @brief   Compute particle to multipole
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <nbfmm/core.hpp>
#include <nbfmm/utility.hpp>

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Compute particle to multipole weighting
///
/// @param[in]   num_particle     the number of particles.
/// @param[in]   gpuptr_position  the original particle effects.
/// @param[in]   gpuptr_weight    the particle effects.
/// @param[out]  p2m_buffer       the workspace.
///
__global__ void p2m_weighting( int num_particle, float2* gpuptr_position, float* gpuptr_weight, float2* p2m_buffer ) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if ( idx>=num_particle )
  {
   return;
  }
  p2m_buffer[idx].x=gpuptr_position[idx].x*gpuptr_weight[idx];
  p2m_buffer[idx].y=gpuptr_position[idx].y*gpuptr_weight[idx];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Compute particle to multipole averaging
///
/// @param[in]   base_dim              the number of cells in the base level per side.
/// @param[out]  gpuptr_cell_position  the cell positions.
/// @param[in]   gpuptr_cell_weight    the cell positions.
///
__global__ void p2m_averaging( int base_dim, float2* gpuptr_cell_position, float* gpuptr_cell_weight ) {
  int thread2Dpx = threadIdx.x + blockIdx.x * blockDim.x;
  int thread2Dpy = threadIdx.y + blockIdx.y * blockDim.y;
  if (thread2Dpx >= base_dim || thread2Dpy >= base_dim)
   return;
  int thread1Dp = thread2Dpy * base_dim + thread2Dpx;

  gpuptr_cell_position[thread1Dp].x=gpuptr_cell_position[thread1Dp].x/gpuptr_cell_weight[thread1Dp];
  gpuptr_cell_position[thread1Dp].y=gpuptr_cell_position[thread1Dp].y/gpuptr_cell_weight[thread1Dp];
}

//  The namespace NBFMM
namespace nbfmm {

// P2M
void Solver::p2m( const int num_particle ) {
  const dim3 kNumThread_cellwise(32,32,1);
  const dim3 kNumBlock_cellwise(((base_dim_-1)/kNumThread_cellwise.x)+1,((base_dim_-1)/kNumThread_cellwise.y)+1,1);
  const int kNumThread_pointwise = 1024;
  const int kNumBlock_pointwise  = ((num_particle-1)/kNumThread_pointwise)+1;

  float2* p2m_buffer;
  int2* p2m_workingspace;
  hipMalloc(&p2m_buffer,      max_num_particle_ * sizeof(float2));
  hipMalloc(&p2m_workingspace, base_dim_ * base_dim_*sizeof(int2));
  p2m_weighting<<<kNumBlock_pointwise,kNumThread_pointwise>>>(num_particle,gpuptr_position_,gpuptr_weight_,p2m_buffer);

  thrust::device_ptr<int2> thrust_index(gpuptr_index_);
  thrust::device_ptr<float2> thrust_position(gpuptr_position_);
  thrust::device_ptr<float> thrust_weight(gpuptr_weight_);
  thrust::device_ptr<float2> thrust_weighted(p2m_buffer);
  thrust::device_ptr<int2> thrust_working(p2m_workingspace);
  thrust::device_ptr<float2> thrust_cellPos(gpuptr_cell_position_);
  thrust::device_ptr<float> thrust_cellWei(gpuptr_cell_weight_);

  thrust::reduce_by_key(thrust_index, thrust_index + num_particle, thrust_weighted, thrust_working, thrust_cellPos);
  thrust::reduce_by_key(thrust_index, thrust_index + num_particle, thrust_weight, thrust_working, thrust_cellWei);
  p2m_averaging<<<kNumBlock_cellwise,kNumThread_cellwise>>>(base_dim_,gpuptr_cell_position_,gpuptr_cell_weight_);
}

}  // namespace nbfmm
