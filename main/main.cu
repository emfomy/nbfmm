#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    main/main.cu
/// @brief   The main code
///
/// @author  Mu Yang <emfomy@gmail.com>
/// @author  Yung-Kang Lee <blasteg@gmail.com>
///

#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include <iostream>
#include <nbfmm/core.hpp>
#include <nbfmm/utility.hpp>
#include <nbfmm/visualization.hpp>
#include <SyncedMemory.h>

using namespace std;

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Main function
///
int main( int argc, char const *argv[] ) {
  cout << "NBFMM "
       << NBFMM_VERSION_MAJOR << "."
       << NBFMM_VERSION_MINOR << "."
       << NBFMM_VERSION_PATCH << endl << endl;

  const char* result_y4m = ( argc > 1 ) ? argv[1] : "nbfmm_result.mp4";

  const int width        = 1024;
  const int height       = 768;
  const int FPS          = 60;
  const unsigned n_frame = 300;
  const int n_star       = 50000;

  const int num_level        = 4;
  const int max_num_particle = n_star;

  float4 position_limit      = make_float4(0.0f, 0.0f, 64.0f, 48.0f);
  float2 position_center     = make_float2(position_limit.x+position_limit.z,
                                           position_limit.y+position_limit.w)/2;
  float2 position_half_size  = make_float2(position_limit.z-position_limit.x,
                                           position_limit.w-position_limit.y)/2;
  float4 visualization_limit = make_float4(position_center.x-position_half_size.x*0.8,
                                           position_center.y-position_half_size.y*0.8,
                                           position_center.x+position_half_size.x*0.8,
                                           position_center.y+position_half_size.y*0.8);

  Stars asteroids(n_star, FPS);
  asteroids.initialize(position_limit);
  asteroids.deletion_check(position_limit);

  nbfmm::Solver solver(num_level, max_num_particle, position_limit);

  unsigned FRAME_SIZE = width*height*3/2;
  MemoryBuffer<uint8_t> frameb(FRAME_SIZE);
  auto frames = frameb.CreateSync(FRAME_SIZE);
  FILE *fp = fopen(result_y4m, "wb");
  fprintf(fp, "YUV4MPEG2 W%d H%d F%d:%d Ip A1:1 C420\n", width, height, FPS, 1);

  int progress = 0;
  printf("=>");

  for (unsigned j = 0; j < n_frame; ++j) {
    fputs("FRAME\n", fp);
    asteroids.visualize(width, height,frames.get_gpu_wo(),1,visualization_limit);
    fwrite(frames.get_cpu_ro(), sizeof(uint8_t), FRAME_SIZE, fp);

    solver.solve(asteroids.n_star, asteroids.gpu_star_position_cur, asteroids.gpu_star_weight, asteroids.gpu_star_acceleration);
    asteroids.update();
    asteroids.deletion_check(position_limit);

    if ( 100 * j > n_frame * progress ) {
      ++progress;
      printf("\b=>");
      fflush(stdout);
    }
  }
  printf("\n");

  fclose(fp);

  return 0;
}
