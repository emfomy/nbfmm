#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    main/main.cu
/// @brief   The main code
///
/// @author  Mu Yang <emfomy@gmail.com>
/// @author  Yung-Kang Lee <blasteg@gmail.com>
///

#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include <iostream>
#include <nbfmm/core.hpp>
#include <nbfmm/display.hpp>
#include <nbfmm/model.hpp>
#include <nbfmm/utility.hpp>
#include <SyncedMemory.h>

using namespace std;
using namespace nbfmm;

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Main function
///
int main( int argc, char const *argv[] ) {
  cout << "NBFMM "
       << NBFMM_VERSION_MAJOR << "."
       << NBFMM_VERSION_MINOR << "."
       << NBFMM_VERSION_PATCH << endl << endl;

  const char* result_y4m = ( argc > 1 ) ? argv[1] : "nbfmm.y4m";

  const int width     = 1024;
  const int height    = 768;
  const int fps       = 60;
  const int num_frame = 3000;

  const int   num_star    = 10000;
  const int   fmm_level   = 4;
  const float tick        = 0.05/fps;
  const float grav_const  = 1.0f;
  const float size_scale  = 1.0f;

  const float pos_width     = 16.0f;
  const float pos_height    = 12.0f;
  const float display_scale = 0.8;
  const float4 position_limits = make_float4(-pos_width/2, -pos_height/2, pos_width/2, pos_height/2);
  const float4 display_limits  = make_float4(position_limits.x * display_scale,
                                             position_limits.y * display_scale,
                                             position_limits.z * display_scale,
                                             position_limits.w * display_scale);

  int progress = 0;
  for ( auto i = 0; i < 100; ++i ) {
    putchar('=');
  }
  putchar('\r'); fflush(stdout);

  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  Stars asteroids(fmm_level, num_star, width, height, fps, tick, grav_const, size_scale, position_limits, display_limits);

  const float2 center_position = (make_float2(position_limits.x, position_limits.y) +
                                  make_float2(position_limits.z, position_limits.w)) / 2;
  const float model_width  = (position_limits.z - position_limits.x)/2;
  const float model_height = (position_limits.w - position_limits.y)/2;

  asteroids.initialize(model::generateRectangle,
      num_star, center_position, model_width, model_height, 16.0f
  );

  // const float2 center_position = (make_float2(position_limits.x, position_limits.y) +
  //                                 make_float2(position_limits.z, position_limits.w)) / 2;
  // const float radius = (position_limits.w - position_limits.y)/16;

  // asteroids.initialize(model::generateDisk,
  //     num_star, center_position, radius, 1.0f
  // );

  // const int n1 = 5;
  // const int n2 = 3;
  // const float mu1 = float(n1) / (n1+n2);
  // const float mu2 = float(n2) / (n1+n2);

  // const float2 center_position1 = (make_float2(position_limits.x, position_limits.y) * (3*mu1+2*mu2) +
  //                                  make_float2(position_limits.z, position_limits.w) * (3*mu1+4*mu2)) / 6;
  // const float2 center_position2 = (make_float2(position_limits.z, position_limits.w) * (3*mu2+2*mu1) +
  //                                  make_float2(position_limits.x, position_limits.y) * (3*mu2+4*mu1)) / 6;
  // const float radius = (position_limits.w - position_limits.y)/16;

  // asteroids.initialize(model::generateDoubleDisk,
  //     num_star*mu1, num_star*mu2, center_position1, center_position2, radius*mu1, radius*mu2, 1.0f
  // );

  // asteroids.initialize(model::generateDoubleDiskCenter,
  //     num_star*mu1, num_star*mu2, center_position1, center_position2, radius*mu1, radius*mu2, 1.0f,
  //     num_star*mu1, num_star*mu1
  // );

  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  int FRAME_SIZE = width*height*3/2;
  MemoryBuffer<uint8_t> frameb(FRAME_SIZE);
  auto frames = frameb.CreateSync(FRAME_SIZE);
  FILE *fp = fopen(result_y4m, "wb");
  fprintf(fp, "YUV4MPEG2 W%d H%d F%d:%d Ip A1:1 C420\n", width, height, fps, 1);

  fputs("FRAME\n", fp);
  asteroids.display(frames.get_gpu_wo());
  fwrite(frames.get_cpu_ro(), sizeof(uint8_t), FRAME_SIZE, fp);

  for ( auto j = 1; j < num_frame; ++j) {
    asteroids.update();
    asteroids.display(frames.get_gpu_wo());
    fputs("FRAME\n", fp);
    fwrite(frames.get_cpu_ro(), sizeof(uint8_t), FRAME_SIZE, fp);

    if ( 100 * j > progress * num_frame ) {
      ++progress;
      putchar('>'); fflush(stdout);
    }
  }
  putchar('\n'); putchar('\n');

  fclose(fp);

  return 0;
}
