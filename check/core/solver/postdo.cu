#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    check/core/solver/postdo.cu
/// @brief   Test nbfmm::Solver::postdo
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include "../solver.hpp"

void TestNbfmmSolver::postdo() {
  hipError_t cuda_status;

  // Alias vectors
  auto effect               = random_position;
  auto perm                 = random_perm;
  auto gpuptr_effect_origin = gpuptr_float2;

  // Allocate memory
  float2 effect_origin[num_particle];

  // Copy input vectors
  cuda_status = hipMemcpy(solver.gpuptr_effect_, effect, num_particle * sizeof(float2), hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);
  cuda_status = hipMemcpy(solver.gpuptr_perm_,   perm,   num_particle * sizeof(int),    hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);

  // Run postdo
  solver.postdo(num_particle, gpuptr_effect_origin);

  // Copy output vectors
  cuda_status = hipMemcpy(effect_origin, gpuptr_effect_origin, num_particle * sizeof(float2), hipMemcpyDeviceToHost);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);

  // Check
  for ( auto i = 0; i < num_particle; ++i ) {
    CPPUNIT_ASSERT(effect[i] == effect_origin[perm[i]]);
  }
}
