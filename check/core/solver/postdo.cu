#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    check/core/solver/postdo.cu
/// @brief   Test nbfmm::Solver::postdo
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include "../solver.hpp"
#include <algorithm>
#include <numeric>

using namespace nbfmm;
using namespace std;

void TestNbfmmSolver::postdo() {
  Solver& solver = *ptr_solver;
  hipError_t cuda_status;

  // Allocate memory
  float2 effect_origin[num_particle];
  int    perm[num_particle];

  // Alias vectors
  auto effect               = random_position;
  auto gpuptr_effect_origin = gpuptr_float2;

  // Create random permutation
  iota(perm, perm+num_particle, 0);
  random_shuffle(perm, perm+num_particle);

  // Copy input vectors
  cuda_status = hipMemcpy(solver.gpuptr_effect_, effect, num_particle * sizeof(float2), hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);
  cuda_status = hipMemcpy(solver.gpuptr_perm_,   perm,   num_particle * sizeof(int),    hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);

  // Run postdo
  solver.postdo(num_particle, gpuptr_effect_origin);

  // Copy output vectors
  cuda_status = hipMemcpy(effect_origin, gpuptr_effect_origin, num_particle * sizeof(float2), hipMemcpyDeviceToHost);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);

  // Check
  for ( auto i = 0; i < num_particle; ++i ) {
    CPPUNIT_ASSERT(effect[i] == effect_origin[perm[i]]);
  }
}
