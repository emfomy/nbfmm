#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    check/core/solver/p2p.cu
/// @brief   Test nbfmm::Solver::p2p
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include "../solver.hpp"
#include <algorithm>

using namespace nbfmm;
using namespace std;

void TestNbfmmSolver::p2p() {
  Solver& solver = *ptr_solver;
  hipError_t cuda_status;

  int num_particle = 2;

  // Alias vectors
  auto position = random_uniform2;
  auto weight   = random_exponential;

  // Allocate memory
  float2 effect0[num_particle];
  float2 effect[num_particle];
  int2   index[num_particle];
  int    head[num_cell_p1];

  // Fill index and head
  int num0 = 0;
  int num1 = num_particle;
  int num2 = num_particle;
  int num3 = num_particle;
  int num4 = num_particle;
  head[0]          = num0;
  head[1]          = num1;
  head[base_dim]   = num2;
  head[base_dim+1] = num3;
  fill(head+2,          head+base_dim,    num2);
  fill(head+base_dim+2, head+num_cell_p1, num4);
  fill(index+num0, index+num1, make_int2(0, 0));
  fill(index+num1, index+num2, make_int2(0, 1));
  fill(index+num2, index+num3, make_int2(1, 0));
  fill(index+num3, index+num4, make_int2(1, 1));

  // Compute effects
  #pragma omp for
  for ( auto i = 0; i < num_particle; ++i ) {
    effect0[i] = make_float2(0.0f, 0.0f);
    for ( auto j = 0; j < num_particle; ++j ) {
      if ( i != j ) {
        effect0[i] += kernelFunction(position[i], position[j], weight[j]);
      }
    }
  }

  // Copy input vectors
  cuda_status = hipMemcpy(solver.gpuptr_position_, position, num_particle * sizeof(float2), hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);
  cuda_status = hipMemcpy(solver.gpuptr_weight_,   weight,   num_particle * sizeof(float),  hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);
  cuda_status = hipMemcpy(solver.gpuptr_index_,    index,    num_particle * sizeof(int2),   hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);
  cuda_status = hipMemcpy(solver.gpuptr_head_,     head,     num_cell_p1  * sizeof(int),    hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);

  // Run predo
  solver.p2p(num_particle);

  // Copy output vectors
  cuda_status = hipMemcpy(effect, solver.gpuptr_effect_, num_particle * sizeof(float2), hipMemcpyDeviceToHost);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);

  // Check
  for ( auto i = 0; i < num_particle; ++i ) {
    CPPUNIT_ASSERT(abs(effect[i].x - effect0[i].x) < 1e-4);
    CPPUNIT_ASSERT(abs(effect[i].y - effect0[i].y) < 1e-4);
  }
}
