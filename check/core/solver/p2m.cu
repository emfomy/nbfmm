#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    check/core/solver/p2m.cu
/// @brief   Test nbfmm::Solver::p2m
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include "../solver.hpp"

using namespace nbfmm;
using namespace std;

void TestNbfmmSolver::p2m() {
  hipError_t cuda_status;

  // Alias vectors
  auto position = random_position;
  auto weight   = random_weight;
  auto index    = random_index;
  auto head     = random_head;

  // Allocate memory
  float2 cell_position0[base_dim * base_dim];
  float2 cell_position[base_dim * base_dim];
  float  cell_weight0[base_dim * base_dim];
  float  cell_weight[base_dim * base_dim];

  // Copy input vectors
  cuda_status = hipMemcpy(solver.gpuptr_position_, position, num_particle * sizeof(float2), hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);
  cuda_status = hipMemcpy(solver.gpuptr_weight_,   weight,   num_particle * sizeof(float),  hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);
  cuda_status = hipMemcpy(solver.gpuptr_index_,    index,    num_particle * sizeof(int2),   hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);
  cuda_status = hipMemcpy(solver.gpuptr_head_,     head,     num_cell_p1  * sizeof(int),    hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);

  // Compute cell positions and weights
  for ( auto i = 0; i < base_dim * base_dim; ++i ) {
    cell_position0[i] = make_float2(0, 0);
    cell_weight0[i]   = 0;
    for ( auto idx = head[i]; idx < head[i+1]; ++idx ) {
      cell_position0[i] += position[idx] * weight[idx];
      cell_weight0[i]   += weight[idx];
    }
    if ( head[i] != head[i+1] ) {
      cell_position0[i] /= cell_weight0[i];
    }
  }

  // Run p2m
  solver.p2m(num_particle);

  // Copy output vectors
  cuda_status = hipMemcpy(cell_position, solver.gpuptr_cell_position_,
                           base_dim * base_dim * sizeof(float2), hipMemcpyDeviceToHost);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);
  cuda_status = hipMemcpy(cell_weight,   solver.gpuptr_cell_weight_,
                           base_dim * base_dim * sizeof(float),  hipMemcpyDeviceToHost);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);

  // Check
  for ( auto i = 0; i < base_dim * base_dim; ++i ) {
    // printf("\n (%d, %d): (%12.4e, %12.4e) * %12.4e | (%12.4e, %12.4e) * %12.4e", i % base_dim, i / base_dim,
    //        cell_position0[i].x, cell_position0[i].y, cell_weight0[i], cell_position[i].x, cell_position[i].y, cell_weight[i]);
    CPPUNIT_ASSERT_DOUBLES_EQUAL(cell_position0[i].x, cell_position[i].x, 1e-4);
    CPPUNIT_ASSERT_DOUBLES_EQUAL(cell_position0[i].y, cell_position[i].y, 1e-4);
    CPPUNIT_ASSERT_DOUBLES_EQUAL(cell_weight0[i],     cell_weight[i],     1e-4);
  }
}
