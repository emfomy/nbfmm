#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// @file    check/core/solver/l2l.cu
/// @brief   Test nbfmm::Solver::l2l
///
/// @author  Mu Yang <emfomy@gmail.com>
///

#include "../solver.hpp"
#include <algorithm>
#include <random>

using namespace std;
using namespace nbfmm;

void TestNbfmmSolver::l2l() {
  Solver& solver = *ptr_solver;
  hipError_t cuda_status;

  // Allocate memory
  float2 cell_effect0[num_level][base_dim][base_dim];
  float2 cell_effect[base_dim][base_dim];

  // Generate cell position and weight
  default_random_engine generator;
  uniform_real_distribution<float> rand_position(-1.0, 1.0);
  for ( auto l = 0; l < num_level; ++l ) {
    int cell_size = 1 << l;
    for ( auto j = 0; j < base_dim; j += cell_size ) {
      for ( auto i = 0; i < base_dim; i += cell_size ) {
        cell_effect0[l][j][i].x = rand_position(generator);
        cell_effect0[l][j][i].y = rand_position(generator);
      }
    }
  }

  // Copy input vectors
  cuda_status = hipMemcpy(solver.gpuptr_cell_effect_, cell_effect0,
                           base_dim * base_dim * num_level * sizeof(float2),  hipMemcpyHostToDevice);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);

  // Compute effects
  for ( auto l = num_level-1; l > 0; --l ) {
    int cell_size = 1 << l;
    int shift = cell_size / 2;
    #pragma omp parallel for collapse(2)
    for ( auto j = 0; j < base_dim; j += cell_size ) {
      for ( auto i = 0; i < base_dim; i += cell_size ) {
        cell_effect0[l-1][j][i]             += cell_effect0[l][j][i];
        cell_effect0[l-1][j][i+shift]       += cell_effect0[l][j][i];
        cell_effect0[l-1][j+shift][i]       += cell_effect0[l][j][i];
        cell_effect0[l-1][j+shift][i+shift] += cell_effect0[l][j][i];
      }
    }
  }

  // Run l2l
  solver.l2l();

  // Copy output vectors
  cuda_status = hipMemcpy(cell_effect, solver.gpuptr_cell_effect_,
                           base_dim * base_dim * sizeof(float2), hipMemcpyDeviceToHost);
  CPPUNIT_ASSERT(cuda_status == hipSuccess);
  for ( auto j = 0; j < base_dim; ++j ) {
    for ( auto i = 0; i < base_dim; ++i ) {
      CPPUNIT_ASSERT_DOUBLES_EQUAL(cell_effect0[0][j][i].x, cell_effect[j][i].x, 1e-4);
      CPPUNIT_ASSERT_DOUBLES_EQUAL(cell_effect0[0][j][i].y, cell_effect[j][i].y, 1e-4);
    }
  }
}
